#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int main() {
    hipError_t code=hipInit(0);

    int driverVersion;
    int runtimeVersion;

    // 获取CUDA驱动版本
    hipError_t res = hipDriverGetVersion(&driverVersion);
    if (res != hipSuccess) {
        printf("Error getting CUDA driver version: %d\n", res);
    } else {
        printf("CUDA Driver Version: %d\n", driverVersion);
    }

    // 获取CUDA运行时版本
    hipError_t err = hipRuntimeGetVersion(&runtimeVersion);
    if (err != hipSuccess) {
        printf("Error getting CUDA runtime version: %s\n", hipGetErrorString(err));
    } else {
        printf("CUDA Runtime Version: %d\n", runtimeVersion);
    }

    return 0;
}